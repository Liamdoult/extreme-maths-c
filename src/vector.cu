#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#if !defined(GPU)
#error Please add -DGPU flag to build with GPU support
#endif
#include "extreme_maths.h"


__global__ void _vector_add_float(float *out, float *a, float *b) {
    out[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__global__ void _vector_iadd_float(float *a, float *b) {
    a[blockIdx.x] += b[blockIdx.x];
}

__global__ void _vector_sub_float(float *out, float *a, float *b) {
    out[blockIdx.x] = a[blockIdx.x] - b[blockIdx.x];
}

__global__ void _vector_isub_float(float *a, float *b) {
    a[blockIdx.x] -= b[blockIdx.x];
}

__global__ void _vector_mul_float(float *out, float *a, float *b) {
    out[blockIdx.x] = a[blockIdx.x] * b[blockIdx.x];
}

__global__ void _vector_imul_float(float *a, float *b) {
    a[blockIdx.x] *= b[blockIdx.x];
}

__global__ void _vector_div_float(float *out, float *a, float *b) {
    out[blockIdx.x] = a[blockIdx.x] / b[blockIdx.x];
}

__global__ void _vector_idiv_float(float *a, float *b) {
    a[blockIdx.x] /= b[blockIdx.x];
}

extern "C" {

    void init() {}
    void clean() {}

    struct Vector_float Vector_generate_float(int size) {
        float *array;
        hipMalloc((void**)&array, sizeof(float) * size);

        struct Vector_float vec;
        vec.size = size;
        vec.array = array;
        return vec;
    }

    struct Vector_float Vector_create_float(float *a, int size) {
        Vector_float vec = Vector_generate_float(size);

        hipMemcpy(vec.array, a, sizeof(float) * size, hipMemcpyHostToDevice);

        return vec;
    }


    void Vector_clean_float(struct Vector_float a) {
        hipFree(a.array);
    }


    float* Vector_get_result_float(struct Vector_float *a) {
        float *out;

        out = (float*)malloc(sizeof(float) * a->size);

        hipMemcpy(out, a->array, sizeof(float) * a->size, hipMemcpyDeviceToHost);

        hipError_t err = hipGetLastError();  // add
        if (err != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(err));
        }

        return out;
    }

    void Vector_iadd_float(struct Vector_float *a, struct Vector_float *b) {
        _vector_iadd_float<<<a->size, 1>>>(a->array, b->array);
    }

    void Vector_isub_float(struct Vector_float *a, struct Vector_float *b) {
        _vector_isub_float<<<a->size, 1>>>(a->array, b->array);
    }

    void Vector_imul_float(struct Vector_float *a, struct Vector_float *b) {
        _vector_imul_float<<<a->size, 1>>>(a->array, b->array);
    }

    void Vector_idiv_float(struct Vector_float *a, struct Vector_float *b) {
        _vector_idiv_float<<<a->size, 1>>>(a->array, b->array);
    }

    struct Vector_float Vector_add_float(struct Vector_float *a, struct Vector_float *b) {
        Vector_float out = Vector_generate_float(a->size);

        _vector_add_float<<<a->size, 1>>>(out.array, a->array, b->array);

        return out;
    }

    struct Vector_float Vector_sub_float(struct Vector_float *a, struct Vector_float *b) {
        Vector_float out = Vector_generate_float(a->size);

        _vector_sub_float<<<a->size, 1>>>(out.array, a->array, b->array);

        return out;
    }

    struct Vector_float Vector_mul_float(struct Vector_float *a, struct Vector_float *b) {
        Vector_float out = Vector_generate_float(a->size);

        _vector_mul_float<<<a->size, 1>>>(out.array, a->array, b->array);

        return out;
    }

    struct Vector_float Vector_div_float(struct Vector_float *a, struct Vector_float *b) {
        Vector_float out = Vector_generate_float(a->size);

        _vector_div_float<<<a->size, 1>>>(out.array, a->array, b->array);

        return out;
    }

}
