#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "extreme_maths.h"


__global__ void _vector_add(float *out, float *a, float *b) {
    out[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__global__ void _vector_iadd(float *a, float *b) {
    a[blockIdx.x] += b[blockIdx.x];
}

__global__ void _vector_sub(float *out, float *a, float *b) {
    out[blockIdx.x] = a[blockIdx.x] - b[blockIdx.x];
}

__global__ void _vector_isub(float *a, float *b) {
    a[blockIdx.x] -= b[blockIdx.x];
}

__global__ void _vector_mul(float *out, float *a, float *b) {
    out[blockIdx.x] = a[blockIdx.x] * b[blockIdx.x];
}

__global__ void _vector_imul(float *a, float *b) {
    a[blockIdx.x] *= b[blockIdx.x];
}

__global__ void _vector_div(float *out, float *a, float *b) {
    out[blockIdx.x] = a[blockIdx.x] / b[blockIdx.x];
}

__global__ void _vector_idiv(float *a, float *b) {
    a[blockIdx.x] /= b[blockIdx.x];
}

extern "C" {
     
    struct Vector create_vector(float *a, int size) {
        float *array;

        hipMalloc((void**)&array, sizeof(float) * size);
        hipMemcpy(array, a, sizeof(float) * size, hipMemcpyHostToDevice);

        struct Vector vec;
        vec.size = size;
        vec.array = array;
        return vec;
    }

    void clean_vector(struct Vector a) {
        hipFree(a.array);
    }


    float* get_result(struct Vector *a) {
        float *out;

        out = (float*)malloc(sizeof(float) * a->size);

        hipMemcpy(out, a->array, sizeof(float) * a->size, hipMemcpyDeviceToHost);

        hipError_t err = hipGetLastError();  // add
        if (err != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(err));
        }

        return out;
    }

    void vector_iadd(struct Vector *a, struct Vector *b) { _vector_iadd<<<a->size, 1>>>(a->array, b->array);
    }

    struct Vector vector_add(struct Vector *a, struct Vector *b) {
        float *array;

        hipMalloc((void**)&array, sizeof(float) * a->size);

        _vector_add<<<a->size, 1>>>(array, a->array, b->array);

        struct Vector vec;
        vec.size = a->size;
        vec.array = array;
        return vec;

    }

    void vector_isub(struct Vector *a, struct Vector *b) {
        _vector_isub<<<a->size, 1>>>(a->array, b->array);
    }

    struct Vector vector_sub(struct Vector *a, struct Vector *b) {
        float *array;

        hipMalloc((void**)&array, sizeof(float) * a->size);

        _vector_sub<<<a->size, 1>>>(array, a->array, b->array);

        struct Vector vec;
        vec.size = a->size;
        vec.array = array;
        return vec;

    }

    void vector_imul(struct Vector *a, struct Vector *b) {
        _vector_imul<<<a->size, 1>>>(a->array, b->array);
    }

    struct Vector vector_mul(struct Vector *a, struct Vector *b) {
        float *array;

        hipMalloc((void**)&array, sizeof(float) * a->size);

        _vector_mul<<<a->size, 1>>>(array, a->array, b->array);

        struct Vector vec;
        vec.size = a->size;
        vec.array = array;
        return vec;

    }

    void vector_idiv(struct Vector *a, struct Vector *b) {
        _vector_idiv<<<a->size, 1>>>(a->array, b->array);
    }

    struct Vector vector_div(struct Vector *a, struct Vector *b) {
        float *array;

        hipMalloc((void**)&array, sizeof(float) * a->size);

        _vector_div<<<a->size, 1>>>(array, a->array, b->array);

        struct Vector vec;
        vec.size = a->size;
        vec.array = array;
        return vec;

    }


}
